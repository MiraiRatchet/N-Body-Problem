#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <string>
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <random>
#include <iomanip>

using namespace std;

typedef double  T;

#define MAX(a, b) ((a) > (b)) ? (a) : (b)
#define G 6.67e-11f
#define eps 1e-5f
#define BS 32

//__device__ const bool writeTrajFiles = true;

std::random_device rd;
std::mt19937 gen(rd());

#define MAX_VAL 1e13
std::uniform_real_distribution<> distribution(0, MAX_VAL);

__host__ T fRand(T fMin, T fMax)
{
    T f = distribution(gen) / MAX_VAL;
    return fMin + f * (fMax - fMin);
}

struct point
{
    T mass; 
    T coord[3];
    T vel[3];

__device__ __host__ point& operator=(const point& p)
    {
        mass = p.mass;
        for (int i = 0; i < 3; ++i)
        {
            coord[i] = p.coord[i];
            vel[i] = p.vel[i];
        }
        return *this;
    }
};

ostream &operator<<(std::ostream &out, const point &point)
{
    out << point.coord[0] << " " << point.coord[1] << " " << point.coord[2];
    return out;
}

__host__ int getNum(const char *const fileName)
{
    int n;
    ifstream ifile;
    ifile.open(fileName);
    if (!ifile.is_open())
    {
        cerr << " Error : file with settings is not open !\n";
        return -1;
    }
    ifile >> n;
    ifile.close();
    return n;
}

__host__ int readFile(const int n, const string fileName, point *points)
{
    ifstream ifile;
    ifile.open(fileName);
    string str;
    if (!ifile.is_open())
    {
        cerr << " Error : file with settings is not open !\n";
        return -1;
    }
    getline(ifile, str);
    for (int i = 0; i < n; ++i)
    {
        ifile >> points[i].mass;
        for (int j = 0; j < 3; ++j)
            ifile >> points[i].coord[j];

        for (int j = 0; j < 3; ++j)
            ifile >> points[i].vel[j];
    }
    ifile.close();
    return 0;
}

__host__ int createRandomFile(const int n, const string fileName)
{
    ofstream ofile;
    ofile.open(fileName);
    string str;
    if (!ofile.is_open())
    {
        cerr << " Error : file with settings is not open !\n";
        return -1;
    }
    ofile << n << endl;
    for (int i = 0; i < n; ++i)
    {
        ofile << fRand(9e9, 10e9) << " "; //mass
        for (int j = 0; j < 3; ++j)
            ofile << fRand(-3, 3) << " "; //coord
        for (int j = 0; j < 3; ++j)
            ofile << fRand(-0.3, 0.3) << " "; //vel
        ofile << endl;
    }
    ofile.close();
    return 0;
}

__device__ T *f(const point &p, T *accel, T *result)
{
    //y = {rx,ry,rz,vx,vy,vz}
        result[0] = p.vel[0];
        result[1] = p.vel[1];
        result[2] = p.vel[2];
        result[3] = accel[0];
        result[4] = accel[1];
        result[5] = accel[2];

        return result;
}


__device__ T dnorm(const T *vec1, const T *vec2)
{
    T sum = 0;
    for (int i = 0; i < 3; ++i)
    {
        sum += (vec1[i] - vec2[i])*(vec1[i] - vec2[i]);
    }
    return  sqrtf(sum);//__fsqrt_rn(sum);
}

__device__ void calculateAccelerations(int *N, point * points, T *accel, int id)
{
        point p = points[id];
        for (int dim = 0; dim < 3; ++dim)
        {
            T a = 0;
            for (int i = 0; i < (*N); ++i)
            {
                T k = dnorm(p.coord, points[i].coord);
                T denominator = k*k*k;
                a += points[i].mass * (p.coord[dim] - points[i].coord[dim]) / (MAX(denominator, eps));
            }

            accel[dim] =  -G * a;
        }
        
}

__global__ void simulate(int *N, point* points)
{
    int myid = blockIdx.x * blockDim.x + threadIdx.x;

    const T tau = 1e-3f;
const T tmax=tau;
    T* k1 = new T[6];
    T* k2 = new T[6];
    
    T* accel = new T[3];

    point ps1;
    point p = points[myid];

    const T tau05 = tau / 2;
    const T tlim = tmax - tau05;

    
    if (myid< (*N)) {

    for (T t = 0; t < tlim; t += tau)
    {
        calculateAccelerations(N, points, accel, myid);
        
        f(p, accel, k1);

            ps1 = p;

            for (int j = 0; j < 3; ++j) {                   
                p.coord[j] = ps1.coord[j] + tau05 * k1[j];
                p.vel[j] = ps1.vel[j] + tau05 * k1[3 + j];
            }

            points[myid]=p;

            __syncthreads();
  

        calculateAccelerations(N, points, accel, myid);

        f(p, accel, k2);

            for (int j = 0; j < 3; ++j) {
                p.coord[j] = ps1.coord[j] + tau * k2[j];
                p.vel[j] = ps1.vel[j] + tau * k2[3 + j];
            }

            points[myid]=p;

            __syncthreads();
    }

}
    delete[] accel;    
    delete[] k1;
    delete[] k2;
}

__host__ T getError(const int N, const point* points)
{
    T t = 0;
    point p, my_p;
    
    T error = 0, max_error =0;
    for (int i = 0; i < N; ++i)
    {
        ifstream ifile;
        ifile.open("traj" + to_string(i + 1) + ".txt");
        int count = 0;
        while (count < 200)
        {
            ifile >> t >> p.coord[0] >> p.coord[1] >> p.coord[2];
            ++count;
        }
        my_p = points[i];
        ifile >> t >> p.coord[0] >> p.coord[1] >> p.coord[2];
        error = MAX(MAX(fabs(p.coord[0] - my_p.coord[0]), fabs(p.coord[1] - my_p.coord[1])), fabs(p.coord[2] - my_p.coord[2]));
        if (error>max_error)
        {
        max_error = error;
        }
        
        ifile.close();
    }
    return max_error;
}

int main()
{
    const bool createNewFile = false;

    if (createNewFile)
    {
        createRandomFile(20000, "Nbody.txt");
    }

    const char *filename = "Nbody.txt"; //4body.txt or Nbody.txt

    int N = getNum(filename);
    int *N_dev;
    hipMalloc((void**)&N_dev,sizeof(int));
    hipMemcpy(N_dev,&N,sizeof(int),hipMemcpyHostToDevice);
    point *points = new point[N];

    readFile(N, filename, points);

    point *GPUpoints;
    hipMalloc((void**)&GPUpoints,N*sizeof(point));
    hipMemcpy(GPUpoints,points,N*sizeof(point),hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    if (N < BS)
    {
        simulate<<<1, N>>>(N_dev, GPUpoints);
    } 
    else
    {
        int blocks=N/BS + (N%BS!=0);
        simulate<<<blocks,BS>>>(N_dev, GPUpoints);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);

    printf("Time spent by GPU: %.2f ms\n",elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipMemcpy(points,GPUpoints,N*sizeof(point),hipMemcpyHostToDevice);
    if (filename == "4body.txt")
    {
        cout << "error = " << getError(4,points);
    }


    hipFree(GPUpoints);
    hipFree(N_dev);
    delete[]points;

    return 0;   
}